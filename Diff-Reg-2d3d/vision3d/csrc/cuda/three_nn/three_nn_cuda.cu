#include "three_nn_cuda.cuh"

namespace vision3d {

void three_nn_cuda_launcher(
    const at::Tensor& q_points, const at::Tensor& s_points, at::Tensor& tnn_distances, at::Tensor& tnn_indices) {
  CHECK_CUDA_AND_CONTIGUOUS(q_points);
  CHECK_CUDA_AND_CONTIGUOUS(s_points);
  CHECK_CUDA_AND_CONTIGUOUS(tnn_distances);
  CHECK_CUDA_AND_CONTIGUOUS(tnn_indices);
  CHECK_SCALAR_TYPE_LONG(tnn_indices);

  at::cuda::CUDAGuard device_guard(q_points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int batch_size = q_points.size(0);
  int num_q_points = q_points.size(1);
  int num_s_points = s_points.size(1);

  dim3 grid_dim(GET_BLOCKS(num_q_points, THREADS_PER_BLOCK), batch_size);
  dim3 block_dim(THREADS_PER_BLOCK);

  AT_DISPATCH_FLOATING_TYPES(q_points.scalar_type(), "three_nn_kernel", [&] {
    three_nn_kernel<scalar_t><<<grid_dim, block_dim, 0, stream>>>(
        batch_size,
        num_q_points,
        num_s_points,
        q_points.data_ptr<scalar_t>(),
        s_points.data_ptr<scalar_t>(),
        tnn_distances.data_ptr<scalar_t>(),
        tnn_indices.data_ptr<long>());
  });

  AT_CUDA_CHECK(hipGetLastError());
}

}  // namespace vision3d
